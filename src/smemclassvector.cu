#include "hip/hip_runtime.h"
#include "common.h"
#include "grid.h"
#include "timer.h"

#include <iostream>


/// Instantiate global objects
Mesh u(NX,NY,NZ,3);
Mesh du(NX,NY,NZ,3);
Grid grid(NX,NY,NZ,0.0,2*M_PI);
Timer timer;

__device__ void rollBundleCacheNoShared(Bundle Bndl, Mesh f, const Int i, const Int j, const Int k)
{
	for (Int vi=0;vi<Bndl.nvars_;vi++)
	{
		for (Int q=0;q<4*NG+1;q++)
		{
			Bndl(-2,q,vi) = Bndl(-1,q,vi);
			Bndl(-1,q,vi) = Bndl(0,q,vi);
			Bndl(0,q,vi) = Bndl(1,q,vi);
			Bndl(1,q,vi) = Bndl(2,q,vi);
		}
			

		/// Add last element from shared tile
		Bndl(NG,0,vi) = f(i,j,k,vi);
		Bndl(NG,1,vi) = f(i,j+1,k,vi);
		Bndl(NG,2,vi) = f(i,j,k-1,vi);
		Bndl(NG,3,vi) = f(i,j-1,k,vi);
		Bndl(NG,4,vi) = f(i,j,k+1,vi);
		Bndl(NG,5,vi) = f(i,j+2,k,vi);
		Bndl(NG,6,vi) = f(i,j,k-2,vi);
		Bndl(NG,7,vi) = f(i,j-2,k,vi);
		Bndl(NG,8,vi) = f(i,j,k+2,vi);
	}
}


__device__ void rollBundleCache(Bundle Bndl, Shared fs, const Int lj, const Int lk)
{
	for (Int vi=0;vi<Bndl.nvars_;vi++)
	{
		for (Int q=0;q<4*NG+1;q++)
		{
			Bndl(-2,q,vi) = Bndl(-1,q,vi);
			Bndl(-1,q,vi) = Bndl(0,q,vi);
			Bndl(0,q,vi) = Bndl(1,q,vi);
			Bndl(1,q,vi) = Bndl(2,q,vi);
		}
			

		/// Add last element from shared tile
		Bndl(2,0,vi) = fs(lj,lk,vi);
		Bndl(2,1,vi) = fs(lj+1,lk,vi);
		Bndl(2,2,vi) = fs(lj,lk-1,vi);
		Bndl(2,3,vi) = fs(lj-1,lk,vi);
		Bndl(2,4,vi) = fs(lj,lk+1,vi);
		Bndl(2,5,vi) = fs(lj+2,lk,vi);
		Bndl(2,6,vi) = fs(lj,lk-2,vi);
		Bndl(2,7,vi) = fs(lj-2,lk,vi);
		Bndl(2,8,vi) = fs(lj,lk+2,vi);
	}
}

__global__ void smemClassKernel(Mesh f, Mesh df, Grid grid)
{
	__shared__ Real smem[3*(NY_TILE+2*NG)*(NZ_TILE+2*NG)];

	Shared fs(smem,NY_TILE,NZ_TILE,3,NG); /// Shared memory object for indexing
	
	const Int ng = f.ng_;
	/// Global indices
	const Int j = threadIdx.x + blockIdx.x*blockDim.x;
	const Int k = threadIdx.y + blockIdx.y*blockDim.y;
	
	/// Local indices	
	const Int lj = threadIdx.x;
	const Int lk = threadIdx.y;
	const Int li = 0; /// the "center" of the bundle (fd stencil) in any "roll step".
	                  /// This will always be zero for any
	                  /// global index i along the array.

	/// Bundle memory and Bundle pointer to that memory
	Real vB[3*(4*NG+1)*(1+2*NG)];
	//Real sB[(4*NG+1)*(1+2*NG)];
	Bundle Bndl(&vB[0],4*NG+1,3);

	/// Initialise for rolling cache
	for (Int vi=0;vi<f.nvars_;vi++)
	{
		bundleInit(Bndl,f,j,k,vi);
	}
	__syncthreads();

	const Int vi = 0;
	 
	if (j < f.ny_ && k < f.nz_)
	{
		for (Int i=0;i<f.nx_;i++)
		{
			///Load shared memory and ghostpts
			loadShared(fs,f,
				   i,j,k,
				   lj,lk); //loadShared() def'd in shared.h
			//fs(lk,lj,vi) = f(i+2,j,k);
			__syncthreads();
			
			/// *** ___ Roll the cache ! ___ ***
			/// Load shared tile into local bundle
			//rollBundleCacheNoShared(Bndl,f,i+2,j,k); //Time taken: 0.0583 ms for N=8
			rollBundleCache(Bndl,fs,lj,lk);

			/// Do operations on bundle:	
			df(i,j,k,0) = delz(Bndl,1.0/grid.dx_,li,2);
			       
		}//End for loop over i.
		
	} //End j,k if statement
	
	
}

__host__ void initHost(Mesh &f, const Grid &grid)
{
	Real *x = grid.h_linspace;
	for (Int i=0;i<f.nx_;i++)
	{
		for (Int j=0;j<f.ny_;j++)
		{
			for (Int k=0;k<f.nz_;k++)
			{
				f.h_data[f.indx(i,j,k,2)] = sin(x[k]);
				//f.h_data[f.indx(i,j,k,0)] = f.indx(i,j,k,0);//sin(x[k]);
				//f.h_data[f.indx(i,j,k,1)] = 2*(x[j]+1);
				//f.h_data[f.indx(i,j,k,2)] = 3*(x[k]+1);
			}
		}
	}
}


Int main()
{
	std::cout << "Executing w/ size: (N=" << NN << ")^3" << std::endl;
	u.allocateHost(); u.allocateDevice();
	du.allocateHost(); du.allocateDevice();
	
	grid.setHostLinspace();
	initHost(u,grid);
	//u.print();
	
	timer.createEvents();
	u.copyToDevice();
	dim3 tpb(NY_TILE,NZ_TILE); 
	dim3 blx(NN/NY_TILE,NN/NZ_TILE);
	timer.recordStart();
 
	pbc_x_kernel<<<blx,tpb>>>(u);
	pbc_y_kernel<<<blx,tpb>>>(u);
	pbc_z_kernel<<<blx,tpb>>>(u);

	smemClassKernel<<<blx,tpb>>>(u,du,grid);
	//zderivKernel<<<blx,tpb>>>(u,du,grid.dx_);
//curlKernel<<<blx,tpb>>>(u,du,grid);
	
	timer.recordStop();
	timer.sync();

	du.copyFromDevice();
	printf("%.6f \t %.6f \n",du.h_data[du.indx(0,0,1,0)],du.h_data[du.indx(du.nx_-1,du.ny_-1,du.nz_-1,0)]);
	//du.print();
	
//testCurl(du);
	
	timer.print();
	
	return 0;
};

     
