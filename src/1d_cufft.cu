#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <cmath>

#define NN 8


void init_host(hipfftDoubleComplex *f, double *x)
{
	//f[0].x = 1.0;
	//f[0].y = 0.0;
	
	
	for (int i=0;i<NN;i++)
	{
		f[i].x = cos(x[i]);
		f[i].y = 0.0;
	}
	
}

void printComplex(hipfftDoubleComplex *f)
{
	std::cout << "Printing complex array: \n";
	double re,im;
	
	for (int i=0;i<NN;i++)
	{
		re = f[i].x;
		im = f[i].y;
		
		if (fabs(re) < 1e-14)
			re = 0.0;
		if (fabs(im) < 1e-14)
			im = 0.0;
		
		
		std::cout << "(" << re << "," << im << ") \n";
	}
}

int main()
{
	hipfftHandle plan1d;
	CUFFT_CHECK(hipfftPlan1d(&plan1d,NN,HIPFFT_Z2Z,1));
		    
	hipfftDoubleComplex *h_mem;
	hipfftDoubleComplex *d_in;
	hipfftDoubleComplex *d_out;
	
	cudaCheck(hipHostMalloc((void**)&h_mem,sizeof(hipfftDoubleComplex)*NN));
	cudaCheck(hipMalloc((void**)&d_in,sizeof(hipfftDoubleComplex)*NN));
	cudaCheck(hipMalloc((void**)&d_out,sizeof(hipfftDoubleComplex)*NN));

	double linspace[NN];
	double L0=0.0,L1=2*M_PI;
	double dx = (L1-L0)/NN;
	
	for (int i=0;i<NN;i++)
		linspace[i] = i*dx;
	
	init_host(h_mem,linspace);
	printComplex(h_mem);
	
	cudaCheck(hipMemcpy(d_in,h_mem,sizeof(hipfftDoubleComplex)*NN,hipMemcpyHostToDevice));

	std::cout << "\n Executing forward C2C transform. \n\n";
	if (hipfftExecZ2Z(plan1d,d_in,d_out,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return 0;	
	}
	
	cudaCheck(hipMemcpy(h_mem,d_out,sizeof(hipfftDoubleComplex)*NN,hipMemcpyDeviceToHost));

	printComplex(h_mem);

	std::cout << "\n" << NN << "       " << (NN >> 1) +1 << "\n";
	/// Free mem
	cudaCheck(hipHostFree(h_mem));
	cudaCheck(hipFree(d_in));
	cudaCheck(hipFree(d_out));
	CUFFT_CHECK(hipfftDestroy(plan1d));
	
	return 0;
}
		
