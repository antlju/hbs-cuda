#include "hip/hip_runtime.h"
#include "common.h"
#include "grid.h"
#include "timer.h"

#include <iostream>

#include "bundle.h"
#include "curl_kernel.h"

__host__
void initHost(Mesh &u, const Grid &grid)
{
	const Real *x = grid.h_linspace;
	const Real *y = grid.h_linspace;
	const Real *z = grid.h_linspace;
	
	for (Int i=0;i<u.nx_;i++)
	{
		for (Int j=0;j<u.ny_;j++)
		{
			for (Int k=0;k<u.nz_;k++)
			{
				//h[fIdx(i,j,k)] = i+NX*(j+NY*k);
				u.h_data[u.indx(i,j,k,0)] = 1.0*x[i]+1.0*y[j]+1.0*z[k];
				u.h_data[u.indx(i,j,k,1)] = 2.0*x[i]+2.0*y[j]+2.0*z[k];
				u.h_data[u.indx(i,j,k,2)] = 3.0*x[i]+3.0*y[j]+3.0*z[k];
			}
		}
	}
	
}

/// Instantiate global objects
Mesh u(NX,NY,NZ,3);
Mesh du(NX,NY,NZ,3);
Grid grid(NX,NY,NZ,0.0,2*M_PI);
Timer timer;

__host__ void testCurl(Mesh &du)
{
	Real ref[3];
	ref[0] = 1;
	ref[1] = -2;
	ref[2] = 1;

	Real maxE[3];
	
	for (Int i=0;i<du.nx_;i++)
	{
		for (Int j=0;j<du.ny_;j++)
		{
			for (Int k=0;k<du.nz_;k++)
			{
				for (Int vi=0;vi<du.nvars_;vi++)
				{
					Real val = du.h_data[du.indx(i,j,k,vi)];
					Real err = fabs(val-ref[vi]);
					if (err > maxE[vi])
						maxE[vi] = err;
				}
			}
		}
	}

	std::cout << "x max error: " << maxE[0] << std::endl;
	std::cout << "y max error: " << maxE[1] << std::endl;
	std::cout << "z max error: " << maxE[2] << std::endl;
}


Int main()
{
	std::cout << "Executing w/ size: (N=" << NN << ")^3" << std::endl;
	u.allocateHost(); u.allocateDevice();
	du.allocateHost(); du.allocateDevice();

	grid.setHostLinspace();
	initHost(u,grid);
	//u.print();
	
	timer.createEvents();
	u.copyToDevice();
	dim3 tpb(NY_TILE,NZ_TILE); 
	dim3 blx(NN/NY_TILE,NN/NZ_TILE);
	timer.recordStart();
 
	pbc_x_kernel<<<blx,tpb>>>(u);
	pbc_y_kernel<<<blx,tpb>>>(u);
	pbc_z_kernel<<<blx,tpb>>>(u);
	
	curlKernel<<<blx,tpb>>>(u,du,grid);
	
	timer.recordStop();
	timer.synch();

	du.copyFromDevice();

	du.print();
	
	testCurl(du);
	
	timer.print();
	

	

	return 0;
};

     
