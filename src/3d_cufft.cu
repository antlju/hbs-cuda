#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <cmath>

#define NN 8
#define NX 8
#define NY 8
#define NZ 8

__host__ __device__
inline size_t iindx(size_t i, size_t j,size_t k)
{
	return k+NZ*(j+NY*i);
}

void init_host(hipfftDoubleComplex *f, double *x)
{
	//f[0].x = 1.0;
	//f[0].y = 0.0;
	
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				f[iindx(i,j,k)].x = 1.0;
				f[iindx(i,j,k)].y = 0.0;
			}
		}
	}
	
}

void printComplex(hipfftDoubleComplex *f)
{
	std::cout << "Printing complex array: \n";
	double re,im;
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				re = f[iindx(i,j,k)].x;
				im = f[iindx(i,j,k)].y;
				
				if (fabs(re) < 1e-14)
					re = 0.0;
				if (fabs(im) < 1e-14)
					im = 0.0;
		
		
				std::cout << "(" << re << "," << im << ") ";
			}
			std::cout << "\n";
		}
		std::cout << "---- \n";
	}
}

int main()
{
	hipfftHandle plan3d;
	CUFFT_CHECK(hipfftPlan3d(&plan3d,NX,NY,NZ,HIPFFT_Z2Z));
		    
	hipfftDoubleComplex *h_mem;
	hipfftDoubleComplex *d_in;
	hipfftDoubleComplex *d_out;
	
	cudaCheck(hipHostMalloc((void**)&h_mem,sizeof(hipfftDoubleComplex)*NY*NX*NZ));
	cudaCheck(hipMalloc((void**)&d_in,sizeof(hipfftDoubleComplex)*NX*NY*NZ));
	cudaCheck(hipMalloc((void**)&d_out,sizeof(hipfftDoubleComplex)*NX*NY*NZ));

	double linspace[NX];
	double L0=0.0,L1=2*M_PI;
	double dx = (L1-L0)/NX;
	
	for (int i=0;i<NX;i++)
		linspace[i] = i*dx;
	
	init_host(h_mem,linspace);
	printComplex(h_mem);
	
	cudaCheck(hipMemcpy(d_in,h_mem,sizeof(hipfftDoubleComplex)*NX*NY*NZ,hipMemcpyHostToDevice));

	std::cout << "\n Executing forward C2C transform. \n\n";
	if (hipfftExecZ2Z(plan3d,d_in,d_out,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return 0;	
	}
	
	cudaCheck(hipMemcpy(h_mem,d_out,sizeof(hipfftDoubleComplex)*NX*NY*NZ,hipMemcpyDeviceToHost));

	printComplex(h_mem);

	
	/// Free mem
	cudaCheck(hipHostFree(h_mem));
	cudaCheck(hipFree(d_in));
	cudaCheck(hipFree(d_out));
	CUFFT_CHECK(hipfftDestroy(plan3d));
	
	return 0;
}
		
