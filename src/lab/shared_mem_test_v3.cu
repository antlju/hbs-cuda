#include "hip/hip_runtime.h"

#include "typedefs.h"
#include "errcheck.h"
#define NN 128
#define NX NN
#define NY NN
#define NZ NN
#define NG 2
#define NGHOSTS NG
#define BUNDLESIZE (NN+2*NG)*(4*NG+1)
#define NY_TILE 4
#define NZ_TILE 4
#define NX_TILE 64

__host__ void printfield(Real *h_mem,const Int Nsize, const Int Nvars);
__host__ Real fieldsum(Real *h_mem,const Int Nsize, const Int Nvars);

/// This gives the (i,j,k)-coordinate for component vi of a discretised 3D vector field
/// stored as a linear array in memory. It
__host__ __device__ inline size_t fIdx(const Int i, const Int j, const Int k, const Int vi=0)
{
	return vi*(NZ+2*NGHOSTS)*(NY+2*NGHOSTS)*(NX+2*NGHOSTS)
		+(i+NGHOSTS)+(NX+2*NGHOSTS)*((j+NGHOSTS)+(NY+2*NGHOSTS)*(k+NGHOSTS));
}

__host__ __device__ inline size_t bIdx(const Int i, const Int q,const Int b)
{
	return (i+NG)+q*(NN+2*NG)+b*BUNDLESIZE;
}

__global__ void kernel(Real *f, Real *df)
{

	__shared__ Real B[NX_TILE+2*NG][NY_TILE+2*NG][NZ_TILE+2*NG];
	
	Int k = threadIdx.x + blockIdx.x*blockDim.x;
	Int j = threadIdx.y + blockIdx.y*blockDim.y;
	Int i = threadIdx.z;
	
	Int bk = threadIdx.x + NG;
	Int bj = threadIdx.y + NG;

	Int p = NN/NX_TILE;
	Int pi = 0;
	if (j < NN && k < NN)
	{
		while (pi < p)
		{
				B[i][bj][bk] = f[fIdx(i+pi*NX_TILE,j,k)];
				__syncthreads();
				
				df[fIdx(i+pi*NX_TILE,j,k)] = B[i][bj][bk];

				pi++;
		}
	}
		

}


__host__ void initHost(Real *h)
{
	for (Int i=0;i<NN;i++)
	{
		for (Int j=0;j<NN;j++)
		{
			for (Int k=0;k<NN;k++)
			{
				//h[fIdx(i,j,k)] = i+NX*(j+NY*k);
				h[fIdx(i,j,k)] = 1;
			}
		}
	}
}

Int main()
{
	Int NNG3 = (NN+2*NG)*(NN+2*NG)*(NN+2*NG);
	Real *h_in,*h_out,*d_in,*d_out;
	cudaCheck(hipHostMalloc(&h_in,sizeof(Real)*NNG3));
	cudaCheck(hipHostMalloc(&h_out,sizeof(Real)*NNG3));
	cudaCheck(hipMalloc((void**)&d_in,sizeof(Real)*NNG3));
	cudaCheck(hipMalloc((void**)&d_out,sizeof(Real)*NNG3));

	initHost(h_in);
	//printfield(h_in,NN,1);
	printf("host init sum: %f\n", fieldsum(h_in,NN,1));
	
	cudaCheck(hipMemcpy(d_in,h_in,sizeof(Real)*NNG3,hipMemcpyHostToDevice));
	//const Int tpb = 256;
	//dim3 threadsPerBlock(tpb);
	//const Int blcks = (NN+tpb-1)/tpb;
	//dim3 blocks((NN+tpb+1)/tpb);

	//dim3 tpb(NY_TILE,NZ_TILE); //1024 max threads per block on Quadro P4000
	dim3 tpb(NZ_TILE,NY_TILE,NX_TILE); 
	dim3 blx(NN/NZ_TILE,NN/NY_TILE);
	kernel<<<blx,tpb>>>(d_in,d_out);
	
	cudaCheck(hipMemcpy(h_out,d_out,sizeof(Real)*NNG3,hipMemcpyDeviceToHost));

	//printfield(h_out,NN,1);
	printf("host out sum: %f\n", fieldsum(h_out,NN,1));
	
	cudaCheck(hipHostFree(h_in));
	cudaCheck(hipHostFree(h_out));
	cudaCheck(hipFree(d_in));
	cudaCheck(hipFree(d_out));

	
	return 0;
}

__host__ void printfield(Real *h_mem,const Int Nsize, const Int Nvars)
{
	printf("\n");
	for (Int vi=0;vi<Nvars;vi++)
	{
		printf("---------------- COMPONENT %i --------- \n", vi);
		for (Int i=0;i<Nsize;i++)
		{
			for (Int j=0;j<Nsize;j++)
			{
				for (Int k=0;k<Nsize;k++)
				{
					printf("%.f ", h_mem[fIdx(i,j,k,vi)]);
				}
				printf("\n");
			}
			printf("----------------\n");
		}

	}
	printf("\n");
}

__host__ Real fieldsum(Real *h_mem,const Int Nsize, const Int Nvars)
{

	Real sum = 0;
	for (Int vi=0;vi<Nvars;vi++)
	{
		for (Int i=0;i<Nsize;i++)
		{
			for (Int j=0;j<Nsize;j++)
			{
				for (Int k=0;k<Nsize;k++)
				{
					sum += h_mem[fIdx(i,j,k,vi)];
				}
			
			}
		}

	}
	return sum/(NN*NN*NN);
}

/*
	const Int threadsPerDim = 32; //
	dim3 blockSize(threadsPerDim,threadsPerDim); //Number of threads per block
	                                            //(max for gtx 850M: 1024 = 32*32)
	const Int blocksPerDim = ((NN+NG)+threadsPerDim-1)/threadsPerDim;
	dim3 gridSize(blocksPerDim, blocksPerDim); //Number of blocks per grid

	const Int bundleSize = threadsPerDim*BUNDLESIZE;
	const Int pencilLength = NN;
	const Int pencilsPerBlock = 
	kernel<<<gridSize,blockSize,bundleSize>>>(d_in,d_out);
*/
