#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <cmath>

#define NN 8
#define NX 8
#define NY 8

__host__ __device__
inline size_t iindx(size_t i, size_t j)
{
	return j+NY*i;
}

void init_host(hipfftDoubleComplex *f, double *x)
{
	//f[0].x = 1.0;
	//f[0].y = 0.0;
	
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			f[iindx(i,j)].x = sin(x[j]);
			f[i].y = 0.0;
		}
	}
	
}

void printComplex(hipfftDoubleComplex *f)
{
	std::cout << "Printing complex array: \n";
	double re,im;
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			re = f[iindx(i,j)].x;
			im = f[iindx(i,j)].y;
			
			if (fabs(re) < 1e-14)
				re = 0.0;
			if (fabs(im) < 1e-14)
				im = 0.0;
		
		
		std::cout << "(" << re << "," << im << ") ";
		}
		std::cout << "\n";
	}
}

int main()
{
	hipfftHandle plan2d;
	CUFFT_CHECK(hipfftPlan2d(&plan2d,NX,NY,HIPFFT_Z2Z));
		    
	hipfftDoubleComplex *h_mem;
	hipfftDoubleComplex *d_in;
	hipfftDoubleComplex *d_out;
	
	cudaCheck(hipHostMalloc((void**)&h_mem,sizeof(hipfftDoubleComplex)*NY*NX));
	cudaCheck(hipMalloc((void**)&d_in,sizeof(hipfftDoubleComplex)*NX*NY));
	cudaCheck(hipMalloc((void**)&d_out,sizeof(hipfftDoubleComplex)*NX*NY));

	double linspace[NX];
	double L0=0.0,L1=2*M_PI;
	double dx = (L1-L0)/NX;
	
	for (int i=0;i<NN;i++)
		linspace[i] = i*dx;
	
	init_host(h_mem,linspace);
	printComplex(h_mem);
	
	cudaCheck(hipMemcpy(d_in,h_mem,sizeof(hipfftDoubleComplex)*NX*NY,hipMemcpyHostToDevice));

	std::cout << "\n Executing forward C2C transform. \n\n";
	if (hipfftExecZ2Z(plan2d,d_in,d_out,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return 0;	
	}
	
	cudaCheck(hipMemcpy(h_mem,d_out,sizeof(hipfftDoubleComplex)*NX*NY,hipMemcpyDeviceToHost));

	printComplex(h_mem);

	
	/// Free mem
	cudaCheck(hipHostFree(h_mem));
	cudaCheck(hipFree(d_in));
	cudaCheck(hipFree(d_out));
	CUFFT_CHECK(hipfftDestroy(plan2d));
	
	return 0;
}
		
