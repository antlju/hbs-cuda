#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#define NN 512
#define NX NN
#define NY NN
#define NZ NN
#define NZH NZ/2+1
#define NX_TILE 16
#define NY_TILE 16
#define NZ_TILE NN

typedef int Int;
typedef double Real;
typedef double2 Complex;

__host__ __device__
inline size_t cindx(size_t i, size_t j, size_t k)
{
	return k+((NZ >> 1)+1)*(j+NY*i); /// (NZ >> 1) is a bitshift expression equiv to NZ/2
}

__host__ __device__
inline size_t rindx(size_t i, size_t j, size_t k)
{
	return k+NZ*(j+NY*i); /// (NZ >> 1) is a bitshift expression equiv to NZ/2
}


__global__
void freqDiv_kernel(Complex *f, const Real xlen)
{
	Int II,JJ; 
	Int j = threadIdx.y+blockIdx.y*blockDim.y;
	Int i = threadIdx.x+blockIdx.x*blockDim.x;
        Real k1,k2,k3,Pi=M_PI,fac;
	
	//if (i < NX && j < NY && k < ((NZ >> 1) + 1))
	if (i < NX && j < NY)
	{
		if (2*i<NX)
			II = i;
		else
                        II = NX-i;

		if (2*j<NY)
			JJ = j;
		else
			JJ = NX-j;
		
		k2 = 2*Pi*JJ/xlen;
                k1 = 2*Pi*II/xlen;
		for (Int k=0;k<((NZ >> 1)+1);k++)
		{
			k3 = 2*Pi*k/xlen;
			fac = -1.0*(k1*k1+k2*k2+k3*k3);

			if (fabs(fac) < 1e-14)
			{
				f[cindx(i,j,k)].x = 0.0;
				f[cindx(i,j,k)].y = 0.0;	
			}
			else
			{
				f[cindx(i,j,k)].x = f[cindx(i,j,k)].x/fac;
				f[cindx(i,j,k)].y = f[cindx(i,j,k)].y/fac;
			}
		}
	}
}

void printComplex(Complex *f)
{
	for (Int i=0;i<NX;i++)
	{
		for (Int j=0;j<NY;j++)
		{
			for (Int k=0;k<(NZ/2+1);k++)
			{
				std::cout << std::setprecision(3) << f[cindx(i,j,k)].x << "," <<
					f[cindx(i,j,k)].y << " ";
			}
			std::cout << std::endl;
		}
		std::cout << "----- " << std::endl;
	}
}

void printReal(Real *f)
{
	Real norm = NX*NY*NZ;
	for (Int i=0;i<NX;i++)
	{
		for (Int j=0;j<NY;j++)
		{
			for (Int k=0;k<NZ;k++)
			{
				std::cout << std::setprecision(3) << f[rindx(i,j,k)]/norm << " ";
			}
			std::cout << std::endl;
		}
		std::cout << "----- " << std::endl;
	}
}

void initReal(Real *f, Real *x, Real q1, Real q2)
{
	for (Int i=0;i<NX;i++)
	{
		for (Int j=0;j<NY;j++)
		{
			for (Int k=0;k<NZ;k++)
			{
				f[rindx(i,j,k)] = sin(q1*x[k])+sin(q2*x[j]);
				//f[rindx(i,j,k)] = sin(x[k]);
			}
		}
	}
}

void normalise(Real *f)
{
	Real norm = NX*NY*NZ;
	for (Int i=0;i<NX;i++)
	{
		for (Int j=0;j<NY;j++)
		{
			for (Int k=0;k<NZ;k++)
			{
				f[rindx(i,j,k)] /= norm;
			}
		}
	}
}

void testResult(Real *f, Real *x,Real q1,Real q2)
{
	Real maxErr = 0.0,diff=0.0,anltc=0.0;
	for (Int i=0;i<NX;i++)
	{
		for (Int j=0;j<NY;j++)
		{
			for (Int k=0;k<NZ;k++)
			{
				anltc = -1.0*((sin(q1*x[k])/(q1*q1))+(sin(q2*x[j])/(q2*q2)));
				//anltc = -1.0*sin(x[k]);
				diff = fabs(anltc-f[rindx(i,j,k)]);
				if (diff > maxErr)
					maxErr = diff;
			}
		}
	}
	std::cout << "Max error:" << maxErr << std::endl;
}


Int main()
{

	hipfftHandle pland2z,planz2d;
	CUFFT_CHECK(hipfftPlan3d(&pland2z,NX,NY,NZ,HIPFFT_D2Z));
	CUFFT_CHECK(hipfftPlan3d(&planz2d,NX,NY,NZ,HIPFFT_Z2D));
	
	Real *d_real,*h_real;
	cudaCheck(hipHostMalloc((void**)&h_real,sizeof(Real)*NX*NY*NZ));
	cudaCheck(hipMalloc((void**)&d_real,sizeof(Real)*NX*NY*NZ));
	
	Complex *d_xform,*h_xform;
	cudaCheck(hipHostMalloc((void**)&h_xform,sizeof(Complex)*NX*NY*(NZ/2+1)));
	cudaCheck(hipMalloc((void**)&d_xform,sizeof(Complex)*NX*NY*(NZ/2+1)));

	///-----------------
	double linspace[NX];
	double L0=0.0,L1=2*M_PI;
	double dx = (L1-L0)/NX;
	
	for (int i=0;i<NX;i++)
		linspace[i] = i*dx;

	Real q1=1.0;
	Real q2=q1;
	initReal(h_real,linspace,q1,q2);
	cudaCheck(hipMemcpy(d_real,h_real,sizeof(Real)*NX*NY*NZ,hipMemcpyHostToDevice));
	
	dim3 blx(NX/NX_TILE,NY/NY_TILE);
	dim3 tpb(NX_TILE,NY_TILE);
	
	///--------------
	std::cout << "\n Executing forward R2C transform... \n\n";
	if (hipfftExecD2Z(pland2z,d_real,d_xform) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
		return 0;	
	}

	///--------------
	freqDiv_kernel<<<blx,tpb>>>(d_xform,L1-L0);

	///--------------
	std::cout << "\n Executing backward C2R transform... \n\n";
	if (hipfftExecZ2D(planz2d,d_xform,d_real) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecZ2D backward failed");
		return 0;	
	}

	cudaCheck(hipMemcpy(h_real,d_real,sizeof(Real)*NX*NY*NZ,hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(h_xform,d_xform,sizeof(Complex)*NX*NY*(NZ/2+1),hipMemcpyDeviceToHost));

	//printReal(h_real);
	normalise(h_real);
	testResult(h_real,linspace,q1,q2);
	       
	return 0;
}
