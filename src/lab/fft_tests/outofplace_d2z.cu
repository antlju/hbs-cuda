#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <cmath>

#define NN 4
#define NX NN
#define NY NN
#define NZ NN
#define NZH NZ/2+1

__host__ __device__
inline size_t indx(size_t i, size_t j, size_t k)
{
	return k+(NZ)*(j+NY*i);
}

void setComplex(hipfftDoubleComplex *f)
{
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZH;k++)
			{
				f[k+NZH*(j+NY*i)].x = 0.0;
				f[k+NZH*(j+NY*i)].y = 0.0;
				
				/*
				if (re < 1e-14)
					re = 0.0;

				if (im < 1e-14)
					im = 0.0;
				*/
				
			}
			
		}
		
	}

	f[0].x = sqrt(2*M_PI);
	f[0].y = 0.0;
       
}


void printComplex(hipfftDoubleComplex *f)
{
	std::cout << "\n Printing complex array \n" << std::endl;
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZH;k++)
			{
				double re = f[k+NZH*(j+NY*i)].x;
				double im = f[k+NZH*(j+NY*i)].y;
				
				/*
				if (re < 1e-14)
					re = 0.0;

				if (im < 1e-14)
					im = 0.0;
				*/
				std::cout << re << "," << im << " ";
			}
			std::cout << std::endl;
		}
		std::cout << "--- " << std::endl;
	}
       
}

void printReal(hipfftDoubleReal *f)
{
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				double re = f[k+(NZ+2)*(j+NY*i)];
				if (re < 1e-14)
					re = 0.0;
				
				std::cout << re << " ";
			}
			std::cout << std::endl;
		}
		std::cout << "--- " << std::endl;
	}
       
}

void setReal(hipfftDoubleReal *f)
{
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				f[indx(i,j,k)] = 0.0;
			}

		}

	}
	f[indx(0,0,0)] = sqrt(2*M_PI)*1e10;
	
}

int main()
{

	hipfftHandle pland2z, planz2z;
	CUFFT_CHECK(hipfftPlan3d(&pland2z,NX,NY,NZ,HIPFFT_D2Z));
	CUFFT_CHECK(hipfftPlan3d(&planz2z,NX,NY,NZH,HIPFFT_Z2Z));
	
	hipfftDoubleReal *hreal;
	hipfftDoubleReal *dreal;
	hipfftDoubleComplex *dcomplex;
	hipfftDoubleComplex *hcomplex;
	
	cudaCheck(hipHostMalloc((void**)&hreal,NX*NY*NZ*sizeof(hipfftDoubleReal)));
	cudaCheck(hipHostMalloc((void**)&hcomplex,NX*NY*NZH*sizeof(hipfftDoubleComplex)));
	cudaCheck(hipMalloc((void**)&dreal,NX*NY*NZ*sizeof(hipfftDoubleReal)));
	cudaCheck(hipMalloc((void**)&dcomplex,NX*NY*NZH*sizeof(hipfftDoubleComplex)));
	
	//setReal(hreal);
	//cudaCheck(hipMemcpy(dreal,hreal,NX*NY*NZ*sizeof(hipfftDoubleReal),hipMemcpyHostToDevice));

	setComplex(hcomplex);
	printComplex(hcomplex);
	
	/// Exec real-to-complex transform
	/*
	if (hipfftExecD2Z(pland2z,dreal,dcomplex) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
		return 0;	
	}
	*/
	
	/// Exec complex-to-complex transform
	if (hipfftExecZ2Z(planz2z,dcomplex,dcomplex,HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return 0;	
	}

	cudaCheck(hipMemcpy(hcomplex,dcomplex,
			     NX*NY*NZH*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost));
	
	printComplex(hcomplex);

	
	cudaCheck(hipHostFree(hreal));
	cudaCheck(hipHostFree(hcomplex));
	cudaCheck(hipFree(dreal));
	cudaCheck(hipFree(dcomplex));
	CUFFT_CHECK(hipfftDestroy(pland2z));
	
	return 0;
}
