#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"

typedef double Real;

/// Instantiate global objects
Timer timer;
Mesh u(NY,NY,NZ,1);
Mesh du(NX,NY,NZ,1);
Grid grid(NX,NY,NZ,0.0,2*M_PI);

__inline__ __device__
Real warpReduceSum(Real val)
{
	for (Int offset = warpSize/2; offset > 0; offset /=2)
	{
		val += __shfl_down(val,offset);
	}
	return val;
}

__inline__ __device__
Real blockReduceSum(Real val)
{
	static __shared__ Real shared[32]; // Shared memory for 32 partial sums (warpSize is 32)
	Int lane = threadIdx.x % warpSize;
	Int wid = threadIdx.x / warpSize;

	val = warpReduceSum(val); /// Each warp performs partial reduction

	if (lane == 0)
		shared[wid] = val; // Write reduced value to shared memory

	__syncthreads(); /// Wait for all partial reductions.

	/// Read from shared memory only if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

	if (wid == 0)
		val = warpReduceSum(val); /// Final reduce within first warp

	return val;
}

__global__ void deviceReduceKernel(Real *in, Real *out, Int N)
{
	Real sum = 0.0;

	/// Reduce multiple elements per thread via grid striding
	for (Int i= blockIdx.x *blockDim.x + threadIdx.x;
	     i<N;
	     i+=blockDim.x * gridDim.x)
	{
		sum += in[i];
	}
	sum = blockReduceSum(sum);
	if (threadIdx.x == 0)
		out[blockIdx.x] = sum;
}


__host__ void initHost(Mesh &f, const Grid &grid)
{
	Real *x = grid.h_linspace;
	for (Int i=0;i<f.nx_;i++)
	{
		for (Int j=0;j<f.ny_;j++)
		{
			for (Int k=0;k<f.nz_;k++)
			{
				f.h_data[f.indx(i,j,k,0)] = 1.0;
			}
		}
	}

	//f.h_data[f.indx(0,5,6,0)] = 3.0;
}

Int main()
{
	timer.createEvents();
	std::cout << "Executing w/ size: (N=" << NN << ")^3" << std::endl;
	u.allocateHost(); u.allocateDevice();
	du.allocateHost(); du.allocateDevice();
	
	grid.setHostLinspace();
	initHost(u,grid);
	u.copyToDevice();

	const Int threads = 1024;
	const Int NN3 = (NX+2*NGHOSTS)*(NY+2*NGHOSTS)*(NZ+2*NGHOSTS);
	const Int blocks = min((NN3+(threads-1))/threads,1024);
	

	timer.recordStart();
	deviceReduceKernel<<<blocks,threads>>>(u.d_data,du.d_data,NN3);
	deviceReduceKernel<<<1,1024>>>(du.d_data,du.d_data,NN3);
	timer.recordStop();
	timer.sync();
	timer.print();
	du.copyFromDevice();
	std::cout << fabs(du.h_data[0]-(NY*NX*NZ)) << std::endl;
	//du.print();
	
	return 0;
}

     
/*

 */
