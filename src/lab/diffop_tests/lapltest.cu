#include "hip/hip_runtime.h"
#include "common.h"
#include "grid.h"
#include "timer.h"

#include <iostream>


/// Instantiate global objects
Mesh u(NX,NY,NZ,3);
Mesh du(NX,NY,NZ,1);
Grid grid(NX,NY,NZ,0.0,2*M_PI);
Timer timer;



__global__ void laplKernel(Mesh f, Mesh df, Grid grid)
{
	__shared__ Real smem[3*(NY_TILE+2*NG)*(NZ_TILE+2*NG)];

	Shared fs(smem,NY_TILE,NZ_TILE,3,NG); /// Shared memory object for indexing

	const Real invdx = 1.0/grid.dx_;
	const Real invdx2 = invdx*invdx;
	const Int ng = f.ng_;
	/// Global indices
	const Int j = threadIdx.x + blockIdx.x*blockDim.x;
	const Int k = threadIdx.y + blockIdx.y*blockDim.y;
	
	/// Local indices	
	const Int lj = threadIdx.x;
	const Int lk = threadIdx.y;
	const Int li = 0; /// the "center" of the bundle (fd stencil) in any "roll step".
	                  /// This will always be zero for any
	                  /// global index i along the array.

	/// Bundle memory and Bundle pointer to that memory
	Real vB[3*(4*NG+1)*(1+2*NG)];
	//Real sB[(4*NG+1)*(1+2*NG)];
	Bundle Bndl(&vB[0],4*NG+1,3);
	Real P[1]; /// Local scalar "pencil"

	
	/// Initialise for rolling cache
	for (Int vi=0;vi<f.nvars_;vi++)
	{
		bundleInit(Bndl,f,j,k,vi);
	}
	__syncthreads();
       
	
	//Bndl(-1,0,0) = f(-2,j,k,0);
	//Bndl(0,0,0) = f(-1,j,k,0);
	//Bndl(1,0,0) = f(0,j,k,0);
	//Bndl(2,0,0) = f(1,j,k,0);

	const Int vi = 0;
	 
	if (j < f.ny_ && k < f.nz_)
	{
		for (Int i=0;i<f.nx_;i++)
		{

			
			///Load shared memory and ghostpts
			loadShared(fs,f,
				   i,j,k,
				   lj,lk); //loadShared() def'd in shared.h
			//fs(lk,lj,vi) = f(i+2,j,k);
			__syncthreads();
			
			/// *** ___ Roll the cache ! ___ ***
			rollBundleCache(Bndl,fs,lj,lk);
			//Bndl(-2,0,0) = Bndl(-1,0,0);
			//Bndl(-1,0,0) = Bndl(0,0,0);
			//Bndl(0,0,0) = Bndl(1,0,0);
			//Bndl(1,0,0) = Bndl(2,0,0);
			//Bndl(2,0,0) = f(i+2,j,k);
			
			/// Do operations on bundle:
			//curl(Bndl,P,li,invdx,invdx,invdx);
			lapl(Bndl,P,li,invdx2,invdx2,invdx2);
			// Set pencil
			//df(i,j,k,0) = del2z(Bndl,invdx2,li,0);
			//df(i,j,k,0) = dely(Bndl,invdx,li,1);
			//df(i,j,k,0) = fabs(Bndl(0,0,0)-f(i,j,k,0));
			df(i,j,k,0) = P[0];
			//df(i,j,k,0) = 1; df(i,j,k,1) = 2; df(i,j,k,2) = 3;
			//df(i,j,k,0) = delz(Bndl,invdx,li,0);
			       
		}//End for loop over i.
		
	} //End j,k if statement
	
	
}

__host__ void initHost(Mesh &f, const Grid &grid)
{
	Real *x = grid.h_linspace;
	for (Int i=0;i<f.nx_;i++)
	{
		for (Int j=0;j<f.ny_;j++)
		{
			for (Int k=0;k<f.nz_;k++)
			{
				//f.h_data[f.indx(i,j,k,0)] = 1.0*sin(x[i])+2.0*sin(x[j])+3.0*sin(x[k]);
				//f.h_data[f.indx(i,j,k,2)] = sin(x[k]);
				/// Initialises f = (1z,2x,3y) -> curl(f) = (3,1,2)
				f.h_data[f.indx(i,j,k,0)] = sin(x[j]);
				f.h_data[f.indx(i,j,k,1)] = sin(x[j]);
				f.h_data[f.indx(i,j,k,2)] = 0;
			}
		}
	}
}

Int main()
{
	std::cout << "Executing w/ size: (N=" << NN << ")^3" << std::endl;
	u.allocateHost(); u.allocateDevice();
	du.allocateHost(); du.allocateDevice();
	
	grid.setHostLinspace();
	initHost(u,grid);
	//u.print();
	
	timer.createEvents();
	u.copyToDevice();
	dim3 tpb(NY_TILE,NZ_TILE); 
	dim3 blx(NN/NY_TILE,NN/NZ_TILE);
	timer.recordStart();

	
	pbc_x_kernel<<<blx,tpb>>>(u);
	pbc_y_kernel<<<blx,tpb>>>(u);
	pbc_z_kernel<<<blx,tpb>>>(u);
	
	
	laplKernel<<<blx,tpb>>>(u,du,grid);
	//zderivKernel<<<blx,tpb>>>(u,du,grid.dx_);
//curlKernel<<<blx,tpb>>>(u,du,grid);
	
	timer.recordStop();
	timer.sync();

	du.copyFromDevice();
	//printf("%.6f \t %.6f \n",du.h_data[du.indx(0,0,1,0)],du.h_data[du.indx(du.nx_-1,du.ny_-1,du.nz_-1,0)]);
	//du.print();
	
//testCurl(du);
	du.print();
	timer.print();
	
	return 0;
};

     
