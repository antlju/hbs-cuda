#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"

typedef double Real;

/// Instantiate global objects
Timer timer;
Mesh u(NY,NY,NZ,1);
Mesh du(NX,NY,NZ,1);
Grid grid(NX,NY,NZ,0.0,2*M_PI);

__global__
void reduce0(Mesh f, Mesh out)
{
	__shared__ Real smem[NY_TILE][NZ_TILE];
	/// Global indices
	const Int j = threadIdx.x + blockIdx.x*blockDim.x;
	const Int k = threadIdx.y + blockIdx.y*blockDim.y;

	const Int lj = threadIdx.x;
	const Int lk = threadIdx.y;

	if (j < f.ny_ && k < f.nz_)
	{
		for (Int i=0;i<f.nx_;i++)
		{
			/// Load shared mem
			smem[lj][lk] = f(i,j,k);
			
			/// Do reduction in shared mem
			for (size_t s=blockDim.y/2;s>0; s>>=1)
			{
				if (lk < s)
				{
					smem[lj][lk] += smem[lj][lk +s];
				}
				__syncthreads();
			}

			//write result to global mem
			if (lk == 0) out(0,j,blockIdx.y) += smem[lj][0];
		}
	}
	
}

__host__ void initHost(Mesh &f, const Grid &grid)
{
	Real *x = grid.h_linspace;
	for (Int i=0;i<f.nx_;i++)
	{
		for (Int j=0;j<f.ny_;j++)
		{
			for (Int k=0;k<f.nz_;k++)
			{
				f.h_data[f.indx(i,j,k,0)] = 1.0;
			}
		}
	}
}

Int main()
{
	timer.createEvents();
	std::cout << "Executing w/ size: (N=" << NN << ")^3" << std::endl;
	u.allocateHost(); u.allocateDevice();
	du.allocateHost(); du.allocateDevice();
	
	grid.setHostLinspace();
	initHost(u,grid);
	u.copyToDevice();

	dim3 tpb(NY_TILE,NZ_TILE); 
	dim3 blx(NN/NY_TILE,NN/NZ_TILE);

	timer.recordStart();
	reduce0<<<blx,tpb>>>(u,du);
	reduce0<<<blx,tpb>>>(du,du);
	timer.recordStop();
	timer.sync();
	timer.print();
	du.copyFromDevice();
	//std::cout << du.h_data[du.indx(0,0,0)] << " " << NY*NX*NZ << std::endl;
	du.print();
	
	return 0;
}

     
/*
__global__
void reduce0(Mesh f, Mesh out)
{
	__shared__ Real smem[NY_TILE][NZ_TILE];
	/// Global indices
	const Int j = threadIdx.x + blockIdx.x*blockDim.x;
	const Int k = threadIdx.y + blockIdx.y*blockDim.y;

	const Int lj = threadIdx.x;
	const Int lk = threadIdx.y;

	if (j < f.ny_ && k < f.nz_)
	{
		/// Load shared mem
		smem[lj][lk] = f(0,j,k);

		/// Do reduction in shared mem
		for (size_t s=1;s<blockDim.y; s*= 2)
		{
			Int index = 2 * s * lk;
			if (index < blockDim.y)
			{
				smem[lj][index] += smem[lj][index +s];
			}
			__syncthreads();
		}

		//write result to global mem
		if (lk == 0) out(0,j,blockIdx.y) = smem[lj][0];
	}
	
}




__global__
void reduce0(Mesh f, Mesh out)
{
	__shared__ Real smem[NY_TILE][NZ_TILE];
	/// Global indices
	const Int j = threadIdx.x + blockIdx.x*blockDim.x;
	const Int k = threadIdx.y + blockIdx.y*blockDim.y;

	const Int lj = threadIdx.x;
	const Int lk = threadIdx.y;

	if (j < f.ny_ && k < f.nz_)
	{
		/// Load shared mem
		smem[lj][lk] = f(0,j,k);

		/// Do reduction in shared mem
		for (size_t s=1;s<blockDim.y; s*= 2)
		{
			if (lk % (2*s) == 0)
			{
				smem[lj][lk] += smem[lj][lk +s];
			}
			__syncthreads();
		}

		//write result to global mem
		if (lk == 0) out(0,j,blockIdx.y) = smem[lj][0];
	}
	
}














 */
