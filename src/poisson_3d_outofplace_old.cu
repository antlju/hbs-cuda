#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <cmath>

#define NN 8
#define NX NN
#define NY NN
#define NZ NN
#define NZH NZ/2+1
#define NX_TILE NN
#define NY_TILE NN
#define NZ_TILE NN

__host__ __device__
inline size_t iindx(size_t i, size_t j,size_t k)
{
	return k+(NZ)*(j+NY*i);
}

__host__ __device__
inline size_t oindx(size_t i, size_t j,size_t k)
{
	return k+((NZ >> 1) + 1)*(j+NY*i);
}

__global__
void freqDiv_kernel(hipfftDoubleComplex *f, const double xlen)
{
	int k = threadIdx.z;
	int j = threadIdx.y;
	int i = threadIdx.x;

	if (i < NX && j < NY && k < NZ/2+1)
	{
		f[oindx(i,j,k)].x = oindx(i,j,k);
		f[oindx(i,j,k)].y = k;
	}

}

__global__
void freqDiv_kernel2(double *f)
{
	int k = threadIdx.z;
	int j = threadIdx.y;
	int i = threadIdx.x;

	if (i < NX && j < NY && k < NZ)
	{
		f[iindx(i,j,k)] = iindx(i,j,k);
	}

}

void init_real(hipfftDoubleReal *f, double *x)
{

	
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				f[iindx(i,j,k)] = 0.0;
				//f[iindx(i,j,k)] = 0.0;
			}
		}
	}
	//f[0] = 1.0;
	
}


void printComplex(hipfftDoubleComplex *f)
{
	std::cout << "Printing complex array: \n";
	double re,im;
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<(NZ/2+1);k++)
			{
				re = f[oindx(i,j,k)].x;
				im = f[oindx(i,j,k)].y;
		
		
				std::cout << "(" << re << "," << im << ") ";
			}
			std::cout << "\n";
		}
		std::cout << "---- \n";
	}
}

void printReal(hipfftDoubleReal *f)
{
	std::cout << "Printing real array: \n";
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				
				double re = f[iindx(i,j,k)];
				if (fabs(re) < 1e-14)
					re = 0.0;
				std::cout << re <<" ";
			}
			std::cout << "\n";
		}
		std::cout << "---- \n";
	}
}

void printTest(double *f)
{
	std::cout << "Printing real array: \n";
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				std::cout << f[iindx(i,j,k)] <<" ";
			}
			std::cout << "\n";
		}
		std::cout << "---- \n";
	}
}


void normalise(hipfftDoubleReal *f)
{
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				f[iindx(i,j,k)] = f[iindx(i,j,k)]/(NX*NY*NZ);


			}

		}

	}
}

int main()
{
	hipfftHandle plan3d_d2z,plan3d_z2d;
	CUFFT_CHECK(hipfftPlan3d(&plan3d_d2z,NX,NY,NZ,HIPFFT_D2Z));
	CUFFT_CHECK(hipfftPlan3d(&plan3d_z2d,NX,NY,NZ,HIPFFT_Z2D));
	
	hipfftDoubleComplex *h_mem;
	hipfftDoubleComplex *h_out;
	hipfftDoubleReal *h_rout;
	hipfftDoubleComplex *d_in;
	hipfftDoubleComplex *d_out;
	hipfftDoubleReal *d_rout;
	double *d_test,*h_test;
	cudaCheck(hipHostMalloc((void**)&h_test,sizeof(double)*NX*NY*NZ));
	cudaCheck(hipMalloc((void**)&d_test,sizeof(double)*NX*NY*NZ));
	cudaCheck(hipHostMalloc((void**)&h_mem,sizeof(hipfftDoubleComplex)*NY*NX*NZH));
	cudaCheck(hipHostMalloc((void**)&h_out,sizeof(hipfftDoubleComplex)*NY*NX*NZH));
	cudaCheck(hipHostMalloc((void**)&h_rout,sizeof(hipfftDoubleReal)*NY*NX*NZ));
	cudaCheck(hipMalloc((void**)&d_in,sizeof(hipfftDoubleComplex)*NX*NY*NZH));
	cudaCheck(hipMalloc((void**)&d_out,sizeof(hipfftDoubleComplex)*NX*NY*NZH));
	cudaCheck(hipMalloc((void**)&d_rout,sizeof(hipfftDoubleReal)*NX*NY*NZ));
	hipfftDoubleReal *h_real = (hipfftDoubleReal *)h_mem;
	
	double linspace[NX];
	double L0=0.0,L1=2*M_PI;
	double dx = (L1-L0)/NX;
	
	for (int i=0;i<NX;i++)
		linspace[i] = i*dx;
	
	//init_host(h_mem,linspace);
	init_real(h_real,linspace);
	printReal(h_real);
	
	//cudaCheck(hipMemcpy(d_in,h_mem,sizeof(hipfftDoubleComplex)*NX*NY*NZH,hipMemcpyHostToDevice));
	
	/*
	///--------------
	std::cout << "\n Executing forward R2C transform... \n\n";
	if (hipfftExecD2Z(plan3d_d2z,(hipfftDoubleReal *)d_in,d_out) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
		return 0;	
	}
	cudaCheck(hipDeviceSynchronize());
	*/
	
	///--------------
	std::cout << "\n Running freq div kernel... \n\n";
	dim3 blx(NX/NX_TILE,NY/NY_TILE,NZ/NZ_TILE);
	dim3 tpb(NX_TILE,NY_TILE,NZ_TILE);
	//freqDiv_kernel<<<blx,tpb>>>(d_out,L1-L0);
	freqDiv_kernel2<<<blx,tpb>>>(d_test);
	
	cudaCheck(hipDeviceSynchronize());

	/*	
	///--------------
	std::cout << "\n Executing forward C2R transform... \n\n";
	if (hipfftExecZ2D(plan3d_z2d,d_out,d_rout) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecZ2D Backward failed");
		return 0;	
	}
	*/
	cudaCheck(hipDeviceSynchronize());

	cudaCheck(hipMemcpy(h_test,d_test,sizeof(double)*NX*NY*NZ,hipMemcpyDeviceToHost));
	//cudaCheck(hipMemcpy(h_out,d_out,sizeof(hipfftDoubleComplex)*NX*NY*NZH,hipMemcpyDeviceToHost));
	//cudaCheck(hipMemcpy(h_rout,d_rout,sizeof(hipfftDoubleReal)*NX*NY*NZ,hipMemcpyDeviceToHost));
	//hipfftDoubleReal *h_outReal = (hipfftDoubleReal*)h_out;
	//normalise(h_rout);
	//printComplex(h_out);
	//printReal(h_rout);
	printTest(h_test);
	/// Free mem
	cudaCheck(hipHostFree(h_mem));
	cudaCheck(hipHostFree(h_out));
	cudaCheck(hipHostFree(h_rout));
	cudaCheck(hipFree(d_in));
	cudaCheck(hipFree(d_out));
	CUFFT_CHECK(hipfftDestroy(plan3d_d2z));
	CUFFT_CHECK(hipfftDestroy(plan3d_z2d));
	return 0;
}
		
/*
void init_host(hipfftDoubleComplex *f, double *x)
{
	//f[0].x = 1.0;
	//f[0].y = 0.0;
	
	
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				f[oindx(i,j,k)].x = 1.0;
				f[oindx(i,j,k)].y = 0.0;
			}
		}
	}
	
}

 */
