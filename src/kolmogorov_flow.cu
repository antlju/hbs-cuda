#include "hip/hip_runtime.h"
#include "common.h"

/// Include kernels
#include "rhs_kernels.h"
#include "ustar_kernels.h"
#include "pressure_gradpsi_solenoidal.h"
#include "poisson_fft.h"
#include "reductions.h"

/// Global instantiation of data classes.
/// These can be passed to device kernels. They contain pointers to device memory.
Mesh uu(NX,NY,NZ,3); /// Velocity vector field.
Mesh uStar(NX,NY,NZ,3); /// u*, step velocity vector field.
Mesh RHSk(NX,NY,NZ,3); /// RHS^k Runge-Kutta substep vector field
Mesh RHSk_1(NX,NY,NZ,3); /// RHS^(k-1) Runge-Kutta substep vector field
Mesh Pp(NX,NY,NZ,1); /// Pressure scalar field
Mesh Psi(NX,NY,NZ,1); /// \Psi scalar field
Mesh gradPsi(NX,NY,NZ,3); /// \grad{\Psi} vector field.
Mesh verify(NX,NY,NZ,3); /// Vector field to store analytic solution for verification.
Mesh uu_stats(NX,NY,NZ,3); //// Vector field to store statistics by reduction computations!
Grid grid(NX,NY,NZ,0,2*M_PI); ///
	
Complex *fftComplex;
Real *fftReal;

Timer timer;

SolverParams params;
Real *d_umax;
Real *h_umax;

/// GPU kernel call layout.
dim3 ThreadsPerBlock(NY_TILE,NZ_TILE); 
dim3 NoOfBlocks(NN/NY_TILE,NN/NZ_TILE);

/// Forward declarations
__host__ void copyMeshOnDevice(Mesh in, Mesh out);
__host__ void launch_output();
__host__ void free_device_mem();
__host__ void apply_pbc(Mesh f);
__host__ void update_timestep(SolverParams params, const Real dx, const Real umax);
__host__ void RungeKuttaStepping(Mesh u, Mesh ustar, Mesh rhsk, Mesh rhsk_1,
				 Mesh p, Mesh psi, Mesh gradpsi, Mesh stats,
				 Complex *fftcomplex, Real *fftreal,
				 Grid grid, SolverParams params,
				 hipfftHandle pland2z, hipfftHandle planz2d);

/// main()
Int main() 
{
	timer.createEvents();

	launch_output();
	
	/// Create cuFFT plans.
	hipfftHandle planD2Z,planZ2D;
	CUFFT_CHECK(hipfftPlan3d(&planD2Z,NX,NY,NZ,HIPFFT_D2Z));
	CUFFT_CHECK(hipfftPlan3d(&planZ2D,NX,NY,NZ,HIPFFT_Z2D));
	
	/// -------------------------------------
	/// Device memory allocation.
	/// -------------------------------------
	uu.allocateDevice();
	uStar.allocateDevice();
	RHSk.allocateDevice();
	RHSk_1.allocateDevice();
	Pp.allocateDevice();
	Psi.allocateDevice();
	gradPsi.allocateDevice();
	verify.allocateDevice();
	verify.allocateHost();
	uu_stats.allocateDevice();
	
	grid.setHostLinspace(); /// Allocates and sets host linspace (in this case equivalent to NumPy's np.linspace(0,2*Pi,NX))
	grid.copyLinspaceToDevice(); /// Allocates device memory and copies from host
	
	cudaCheck(hipMalloc((void**)&fftReal,sizeof(Real)*NX*NY*NZ));
	cudaCheck(hipMalloc((void**)&fftComplex,sizeof(Complex)*NX*NY*(NZ/2+1)));

	/// Allocate single value variables on host and device
	cudaCheck(hipHostMalloc(&params.h_dt,sizeof(Real)));
	cudaCheck(hipMalloc((void**)&params.d_dt,sizeof(Real)));
	cudaCheck(hipMalloc((void**)&d_umax,sizeof(Real)));
	cudaCheck(hipHostMalloc(&h_umax,sizeof(Real)));

	
	/// -------------------------------------
	/// Set up solver parameters. ::: This should probably be read from a file.
	///---------------------------------------
	params.maxTimesteps = 12000;
	std::cout << "Max timesteps: " << params.maxTimesteps << std::endl;
	params.currentTimestep = 0;
	params.Uchar = 1.0/2;
	params.rho = 1.0;
	params.viscosity = 1.0/20;
	params.kf = 1.0; /// Kolmogorov frequency.
	params.f0 = 1.0/40;
	//params.h_dt[0] = 0.001;
	params.saveinterval = 100;

	/*
	/// Set up initial timestep size based on forcing
	Real forceabs,fmax = 0.0;
	for (size_t i=0;i<NX;i++)
	{
		forceabs = fabs(sin(params.kf*grid.h_linspace[i]));
		if (forceabs > fmax)
			fmax = forceabs;
	}
	*/
	update_timestep(params,grid.dx_,params.Uchar);
	
	
	uu.allocateHost();
	uStar.allocateHost();
	RHSk.allocateHost();
	std::cout << "grid.dx_: " << grid.dx_ << " 2*pi/128: " << 2*M_PI/128 << std::endl;
	/// -------------------------------------
	/// Run solver for the set maximum no. of timesteps.
	///---------------------------------------
	timer.recordStart();
	for (Int timestep = 0;timestep<params.maxTimesteps;timestep++)
	{
		params.currentTimestep = timestep;
		RungeKuttaStepping(uu,uStar,RHSk,RHSk_1,
				   Pp,Psi,gradPsi,uu_stats,
				   fftComplex,fftReal,
				   grid,params,
				   planD2Z,planZ2D);

		//Pp.copyFromDevice();
		//Psi.copyFromDevice();
		
		if(timestep % params.saveinterval == 0)
		{
			uu.copyFromDevice();
			std::cout << "Step: " << timestep << " umax: " << uu.max() << "\n";
			//std::cout << "Step: " << timestep << " pmax: " << Pp.max() << "\n";
			//std::cout << "Step: " << timestep << " psimax: " << Psi.max() << "\n";
		}
	       
	}
	timer.recordStop();
	timer.sync();
	std::cout << "Finished timestepping after " << params.maxTimesteps << " steps." << std::endl;
	timer.print();
	
	//uu.printfirsty();
	//std::cout << "Maximum value of velocity field: " << uu.max() << "\n";
	


	
       	/// Free device memory.
	free_device_mem();
	CUFFT_CHECK(hipfftDestroy(planD2Z));
	CUFFT_CHECK(hipfftDestroy(planZ2D));
	return 0;
}

/// Runge-Kutta stepping. Computes RK3 substeps from k=1 to k=3
__host__
void RungeKuttaStepping(Mesh u, Mesh ustar, Mesh rhsk, Mesh rhsk_1,
			Mesh p, Mesh psi, Mesh gradpsi, Mesh stats,
			Complex *fftcomplex, Real *fftreal,
			Grid grid, SolverParams params,
			hipfftHandle pland2z, hipfftHandle planz2d)
{
        /// From the previous step we have k=0 (time step n) data.
        /// We want to arrive at data for k=3 (time step n+1).
        /// (compare with Rosti & Brandt 2017)


	for (Int k_rk = 1;k_rk<=3;k_rk++)
	{
		
		/// First calculate RHSk = -D_j u_i u_j+(nu/rho)*Lapl(u))+force
                /// Then calc. u* = u+(2*dt*(alpha(k)/rho))*grad(p)
                ///                        +(dt*beta(k))*RHSk+(dt*gamma(k))*RHSk_1
                /// This is all done within the bundle/pencil framework.

		/// First set RHSk_1 to be RHSk from previous step.
		copyMeshOnDevice(rhsk, rhsk_1);
		
		/// Apply PBCS to u and calculate RHS^k
		apply_pbc(u);
		calculate_RHSk_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(u,rhsk,grid,params);
		//rhsk.copyFromDevice();
		
		//std::cout << "Tstep: " << params.currentTimestep << " k_rk: " << k_rk << " rhsk max: " << rhsk.max() << std::endl;
		// If k_rk == 1 update the timestep dt
                if (k_rk == 1 && params.currentTimestep > 1)
		{
			//calc_max(u,stats);
			//cudaCheck(hipMemcpy(h_umax,&stats.d_data[0],sizeof(Real),hipMemcpyDeviceToHost));
                        update_timestep(params,grid.dx_,h_umax[0]);
		}
		
		
		/// Calculate ustar
		calculate_uStar_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(u,rhsk,rhsk_1,p,ustar,
						       params,grid.dx_,k_rk);
		copyMeshOnDevice(ustar,u);
		
		/// Solve the Poisson equation for Psi.
		//apply_pbc(ustar);
		//ustar.copyFromDevice();
		//std::cout << "printing ustar: " << std::endl;
		//ustar.print();
		///Result from kernel below is stored in the Psi array for input to Poisson solver.
		//calc_divergence_uStar_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(ustar,psi,
		//						     params,grid.dx_,k_rk);
	//psi.copyFromDevice();
	//std::cout << "printing psi: " << std::endl;
	//psi.print();
	//Poisson_FFT_solver(psi,fftcomplex,fftreal,grid.xlen,k_rk,pland2z,planz2d);
		
                /// Update pressure, calculate gradient of psi to enforce solenoidal condition
		//update_pressure_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(p,psi);
		//apply_pbc(psi);
		//calc_gradpsi_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(psi,gradpsi,grid.dx_);
		//enforce_solenoidal_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(u,ustar,gradpsi,
		//							  params,k_rk);
		//copyMeshOnDevice(rhsk, u);
		
	}
	//uu.copyFromDevice();
	//std::cout << params.currentTimestep << ": " << uu.max() << std::endl;
	
}

__host__
void apply_pbc(Mesh f)
{
	pbc_x_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(f);
	pbc_y_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(f);
	pbc_z_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(f);
}

__host__
void free_device_mem()
{
	cudaCheck(hipFree(uu.d_data));
	cudaCheck(hipFree(uStar.d_data));
	cudaCheck(hipFree(RHSk.d_data));
	cudaCheck(hipFree(RHSk_1.d_data));
	cudaCheck(hipFree(Pp.d_data));
	cudaCheck(hipFree(Psi.d_data));
	cudaCheck(hipFree(gradPsi.d_data));
	cudaCheck(hipFree(verify.d_data));
	cudaCheck(hipFree(fftComplex));
	cudaCheck(hipFree(fftReal));
	cudaCheck(hipFree(params.d_dt));
	cudaCheck(hipFree(d_umax));
	cudaCheck(hipFree(uu_stats.d_data));
}

__host__
void launch_output()
{
	std::cout << "Launching kolmogorov flow with cubic space size N^3 = NX*NY*NZ = " << NN << "^3." << std::endl;
	std::cout << "Tile sizes are \n X_TILE: " << NX_TILE << ",\n Y_TILE: " << NY_TILE << ",\n Z_TILE: " << NZ_TILE << "." << std::endl;
}

__host__
void
copyMeshOnDevice(Mesh in, Mesh out)
{
	cudaCheck(hipMemcpy(out.d_data,in.d_data,sizeof(Real)*in.totsize_,hipMemcpyDeviceToDevice));
}

__host__
void update_timestep(SolverParams params, const Real dx, const Real umax)
{
	
	//std::cout << umax << std::endl;
	
	Real c1=1.0/3;
	Real c2=c1; //Courant numbers for advection and diffusion respectively
	
	Real nu = params.viscosity,L=dx;
        Real UU;
	//Factor of 1/3 since dx=dy=dz

	UU = params.Uchar;
	Real adv = c1*dx/UU;
	Real diff = c2*pow(L,2)/nu;
	//std::cout << "adv : " << adv << "\t diff: " << diff << std::endl;
	//Set new time step size according to CFL condition
	if (adv < diff)
		params.h_dt[0] = adv;
	else
		params.h_dt[0] = diff;

	if (params.currentTimestep == params.maxTimesteps-1)
		std::cout << adv << " " << diff << std::endl; /// Print for debug
	
	//params.h_dt[0] = 0.00001;
	
	
	//params.dt_copyToDevice();
}
