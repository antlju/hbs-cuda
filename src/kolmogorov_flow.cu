#include "hip/hip_runtime.h"
/* 7 aug 2018, 13:09. 
I have completed the solver and it seems to give a reasonable value for max(u) up to NN=32, but for larger sizes it diverges! Need to fix
*/
/* 2 aug 2018, 15:45.
Implemented divergence of ustar, this is hard to test w/o full solver implementation since divergence is zero out of the box. But it seems to give back numbers in a proper way (not sure if "correct" numbers).
 */

/* 2 aug 2018, 15:00.
Implementation of RHSk copy, RHSk calc, uStar calc kernels. Don't know if they give the right numbers of course 
but they don't give completely unreasonable numbers and everything compiles fine. cuda-memcheck gives no errors.
 */

#include "common.h"

/// Include solver specific kernels
#include "rhs_kernels.h"
#include "ustar_kernels.h"
#include "pressure_gradpsi_solenoidal.h"
#include "poisson_fft.h"

/// Global instantiation of data classes.
/// These can be passed to device kernels. They contain pointers to device memory.
Mesh uu(NX,NY,NZ,3); /// Velocity vector field.
Mesh uStar(NX,NY,NZ,3); /// u*, step velocity vector field.
Mesh RHSk(NX,NY,NZ,3); /// RHS^k Runge-Kutta substep vector field
Mesh RHSk_1(NX,NY,NZ,3); /// RHS^(k-1) Runge-Kutta substep vector field
Mesh Pp(NX,NY,NZ,1); /// Pressure scalar field
Mesh Psi(NX,NY,NZ,1); /// \Psi scalar field
Mesh gradPsi(NX,NY,NZ,3); /// \grad{\Psi} vector field.
Mesh verify(NX,NY,NZ,3); /// Vector field to store analytic solution for verification.
Grid grid(NX,NY,NZ,0,2*M_PI); ///
	
Complex *fftComplex;
Real *fftReal;

Timer timer;

/// GPU kernel call layout.
dim3 ThreadsPerBlock(NY_TILE,NZ_TILE); 
dim3 NoOfBlocks(NN/NY_TILE,NN/NZ_TILE);

/// Forward declarations
__host__ void copyMeshOnDevice(Mesh in, Mesh out);
__host__ void launch_output();
__host__ void free_device_mem();
__host__ void apply_pbc(Mesh f);
__host__ void RungeKuttaStepping(Mesh u, Mesh ustar, Mesh rhsk, Mesh rhsk_1,
				 Mesh p, Mesh psi, Mesh gradpsi, Complex *fftcomplex, Real *fftreal,
				 Grid grid, SolverParams params,
				 hipfftHandle pland2z, hipfftHandle planz2d);

/*
__host__ Real calc_max_uu(Mesh &u)
{
	Real max = 0.0;
	Real val = 0.0;
	for (Int i=0;u.nx_;i++)
	{
		for (Int j=0;u.ny_;j++)
		{
			for (Int k=0;k<u.nz_;k++)
			{
				for (Int vi=0;vi<u.nvars_;vi++)
				{
					val = u.h_data[u.indx(i,j,k,vi)];
					if (fabs(val) > max)
						max = fabs(val);
				}
			}     
		}
	}
	return max;
}
*/

/// main()
Int main() 
{
	timer.createEvents();
	timer.recordStart();
	launch_output();
	
	/// Create cuFFT plans.
	hipfftHandle planD2Z,planZ2D;
	CUFFT_CHECK(hipfftPlan3d(&planD2Z,NX,NY,NZ,HIPFFT_D2Z));
	CUFFT_CHECK(hipfftPlan3d(&planZ2D,NX,NY,NZ,HIPFFT_Z2D));
	
	/// -------------------------------------
	/// Device memory allocation.
	/// -------------------------------------
	uu.allocateDevice();
	uStar.allocateDevice();
	RHSk.allocateDevice();
	RHSk_1.allocateDevice();
	Pp.allocateDevice();
	Psi.allocateDevice();
	gradPsi.allocateDevice();
	verify.allocateDevice();
	verify.allocateHost();
	
	grid.setHostLinspace(); /// Allocates and sets host linspace (in this case equivalent to NumPy's np.linspace(0,2*Pi,NX))
	grid.copyLinspaceToDevice(); /// Allocates device memory and copies from host
	
	cudaCheck(hipMalloc((void**)&fftReal,sizeof(Real)*NX*NY*NZ));
	cudaCheck(hipMalloc((void**)&fftComplex,sizeof(Complex)*NX*NY*(NZ/2+1)));

	/// -------------------------------------
	/// Set up solver parameters. ::: This should probably be read from a file.
	///---------------------------------------
	SolverParams params;
	params.maxTimesteps = 100;
	params.currentTimestep = 0;
	params.Uchar = 1.0/2;
	params.viscosity = 1.0/10;
	params.kf = 1.0; /// Kolmogorov frequency.

	/// -------------------------------------
	/// Run solver for the set maximum no. of timesteps.
	///---------------------------------------
	for (Int timestep = 0;timestep<params.maxTimesteps;timestep++)
	{
		params.currentTimestep = timestep;
		RungeKuttaStepping(uu,uStar,RHSk,RHSk_1,
				   Pp,Psi,gradPsi,fftComplex,fftReal,
				   grid,params,
				   planD2Z,planZ2D);
	}

	std::cout << "Finished timestepping after " << params.maxTimesteps << " steps." << std::endl;
	uu.allocateHost();
	uu.copyFromDevice();
	std::cout << "Maximum value of velocity field: " << uu.max() << "\n";
	
	timer.recordStop();
	timer.sync();
	timer.print();
       	/// Free device memory.
	free_device_mem();
	CUFFT_CHECK(hipfftDestroy(planD2Z));
	CUFFT_CHECK(hipfftDestroy(planZ2D));
	return 0;
}

/// Runge-Kutta stepping. Computes RK3 substeps from k=1 to k=3
__host__
void RungeKuttaStepping(Mesh u, Mesh ustar, Mesh rhsk, Mesh rhsk_1,
			Mesh p, Mesh psi, Mesh gradpsi, Complex *fftcomplex, Real *fftreal,
			Grid grid, SolverParams params,hipfftHandle pland2z, hipfftHandle planz2d)
{  
        /// From the previous step we have k=0 (time step n) data.
        /// We want to arrive at data for k=3 (time step n+1).
        /// (compare with Rosti & Brandt 2017)

	for (Int k_rk = 1;k_rk<=3;k_rk++)
	{
		/// First calculate RHSk = -D_j u_i u_j+(nu/rho)*Lapl(u))+force
                /// Then calc. u* = u+(2*dt*(alpha(k)/rho))*grad(p)
                ///                        +(dt*beta(k))*RHSk+(dt*gamma(k))*RHSk_1
                /// This is all done within the bundle/pencil framework.

		/// First set RHSk_1 to be RHSk from previous step.
		copyMeshOnDevice(rhsk, rhsk_1);
		
		/// Apply PBCS to u and calculate RHS^k
		apply_pbc(u);
		calculate_RHSk_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(u,rhsk,grid,params);

		/// Calculate ustar
		calculate_uStar_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(u,rhsk,rhsk_1,p,ustar,
					      params,grid.dx_,k_rk);

		/// Solve the Poisson equation for Psi.
		apply_pbc(ustar);
		///Result from kernel below is stored in the Psi array for input to Poisson solver.
		calc_divergence_uStar_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(ustar,psi,
									     params,grid.dx_,k_rk);

		Poisson_FFT_solver(psi,fftcomplex,fftreal,grid.xlen,k_rk,pland2z,planz2d);
		
                /// Update pressure, calculate gradient of psi to enforce solenoidal condition
		update_pressure_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(p,psi);
		apply_pbc(psi);
		calc_gradpsi_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(psi,gradpsi,grid.dx_);
		enforce_solenoidal_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(u,ustar,gradpsi,
									  params,k_rk);	
	}

	
}

__host__
void apply_pbc(Mesh f)
{
	pbc_x_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(f);
	pbc_y_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(f);
	pbc_z_kernel<<<NoOfBlocks,ThreadsPerBlock>>>(f);
}

__host__
void free_device_mem()
{
	cudaCheck(hipFree(uu.d_data));
	cudaCheck(hipFree(uStar.d_data));
	cudaCheck(hipFree(RHSk.d_data));
	cudaCheck(hipFree(RHSk_1.d_data));
	cudaCheck(hipFree(Pp.d_data));
	cudaCheck(hipFree(Psi.d_data));
	cudaCheck(hipFree(gradPsi.d_data));
	cudaCheck(hipFree(verify.d_data));
	cudaCheck(hipFree(fftComplex));
	cudaCheck(hipFree(fftReal));
}

__host__
void launch_output()
{
	std::cout << "Launching kolmogorov flow with cubic space size N^3 = NX*NY*NZ = " << NN << "^3." << std::endl;
	std::cout << "Tile sizes are \n X_TILE: " << NX_TILE << ",\n Y_TILE: " << NY_TILE << ",\n Z_TILE: " << NZ_TILE << "." << std::endl;
}

__host__
void
copyMeshOnDevice(Mesh in, Mesh out)
{
	cudaCheck(hipMemcpy(out.d_data,in.d_data,sizeof(Real)*in.totsize_,hipMemcpyDeviceToDevice));
}
