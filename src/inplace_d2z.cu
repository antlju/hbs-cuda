#include <hipfft/hipfft.h>
#include "errcheck.h"
#include "cuffterr.h"
#include <iostream>
#include <cmath>

#define NN 4
#define NX NN
#define NY NN
#define NZ NN
#define NZH NZ/2+1

__host__ __device__
inline size_t oindx(int i, int j, int k)
{
	return k+NZH*(j+NY*i);
}

__host__ __device__
inline size_t iindx(int i, int j, int k)
{
	return k+(NZ+2)*(j+(NY)*i);
}

void inithostmem(double *h_mem)
{
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZ;k++)
			{
				h_mem[k+NY*(j+(NX+2)*i)] = 1.0;
			}
		}
	}
}

void printHost(hipfftDoubleComplex *f)
{
	for (int i=0;i<NX;i++)
	{
		for (int j=0;j<NY;j++)
		{
			for (int k=0;k<NZH;k++)
			{
				double re = f[k+NZH*(j+NY*i)].x;
				double im = f[k+NZH*(j+NY*i)].y;
				std::cout << re << "," << im << " ";
			}
			std::cout << std::endl;
		}
		std::cout << "--- " << std::endl;
	}
       
}

int main()
{
	double *h_mem;
	hipfftDoubleComplex *xform_h_mem;
	cudaCheck(hipHostMalloc(&h_mem,NX*NY*NZ*sizeof(double)));
	cudaCheck(hipHostMalloc(&xform_h_mem,NX*NY*NZH*sizeof(hipfftDoubleComplex)));
	inithostmem(h_mem);
	
	hipfftDoubleComplex *d_mem;

	cudaCheck(hipMalloc(&d_mem,NX*NY*NZH*sizeof(hipfftDoubleComplex)));
	cudaCheck(hipMemcpy(d_mem,h_mem,NX*NY*NZH*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice));

	hipfftHandle pland2z;

	CUFFT_CHECK(hipfftPlan3d(&pland2z,NX,NY,NZ,HIPFFT_D2Z));
	CUFFT_CHECK(hipfftExecD2Z(pland2z,(hipfftDoubleReal*)d_mem,d_mem));

	cudaCheck(hipMemcpy(xform_h_mem,d_mem,
			     NX*NY*NZH*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost));

	printHost(xform_h_mem);
	return 0;
}
